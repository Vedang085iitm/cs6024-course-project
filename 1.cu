#include <bits/stdc++.h>
#include <numeric>
#include <chrono>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <fstream>

using namespace std;

using std::cin;
using std::cout;


__global__ void rowsum(int *d_matrix, int *d_sums, int n)
{
    long block_Idx = blockIdx.x + (gridDim.x) * blockIdx.y + (gridDim.y * gridDim.x) * blockIdx.z;
    long thread_Idx = threadIdx.x + (blockDim.x) * threadIdx.y + (blockDim.y * blockDim.x) * threadIdx.z;
    long block_Capacity = blockDim.x * blockDim.y * blockDim.z;
    long i = block_Idx * block_Capacity + thread_Idx;

    if (i < n)
    {
        d_sums[i] = 0; // Initialize the sum to 0
        for (int j = 0; j < n; ++j)
        {
            atomicAdd(&d_sums[i], d_matrix[i * n + j]);
        }
    }
}

__global__ void neighborJoiningMatrix(int *d_matrix, int *d_rowSums, int *d_njMatrix, int n)
{
    long block_Idx = blockIdx.x + (gridDim.x) * blockIdx.y + (gridDim.y * gridDim.x) * blockIdx.z;
    long thread_Idx = threadIdx.x + (blockDim.x) * threadIdx.y + (blockDim.y * blockDim.x) * threadIdx.z;
    long block_Capacity = blockDim.x * blockDim.y * blockDim.z;
    long arr_Idx = block_Idx * block_Capacity + thread_Idx;

    if (arr_Idx < n * n)
    {
        int i = arr_Idx / n;
        int j = arr_Idx % n;

        if (i == j)
        {
            d_njMatrix[arr_Idx] = 0;
        }
        else
        {
            d_njMatrix[arr_Idx] = (n - 2) * d_matrix[arr_Idx] - d_rowSums[i] - d_rowSums[j];
        }
    }
}

__global__ void findMinIndices(int *d_njMatrix, int *d_rowSums, int *d_minVal, int *d_minIndexI, int *d_minIndexJ, int *d_delta, int n)
{
    extern __shared__ int sharedData[];
    extern __shared__ int sharedIndices[];

    long block_Idx = blockIdx.x + (gridDim.x) * blockIdx.y + (gridDim.y * gridDim.x) * blockIdx.z;
    long thread_Idx = threadIdx.x + (blockDim.x) * threadIdx.y + (blockDim.y * blockDim.x) * threadIdx.z;
    long block_Capacity = blockDim.x * blockDim.y * blockDim.z;
    long arr_Idx = block_Idx * block_Capacity + thread_Idx;

    int i = arr_Idx / n;
    int j = arr_Idx % n;

    if (i < n && j < n && i != j)
    {
        sharedData[thread_Idx] = d_njMatrix[arr_Idx];
        sharedIndices[thread_Idx] = arr_Idx;
    }
    else
    {
        sharedData[thread_Idx] = INT_MAX;
    }

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (thread_Idx < s)
        {
            if (sharedData[thread_Idx] > sharedData[thread_Idx + s])
            {
                sharedData[thread_Idx] = sharedData[thread_Idx + s];
                sharedIndices[thread_Idx] = sharedIndices[thread_Idx + s];
            }
        }
        __syncthreads();
    }
  
    if (thread_Idx == 0)
    {
        *d_minVal = sharedData[0];
     
        *d_minIndexI = sharedIndices[0] / n;
        *d_minIndexJ = sharedIndices[0] % n;
        *d_delta = (d_rowSums[*d_minIndexI] - d_rowSums[*d_minIndexJ]) / (n - 2);
    }
 
__syncthreads();

}

int main(int argc, char **argv)
{

    int n;
    // cout << "Enter the size of the matrix: " << endl;
    cin >> n;
    // cout << "Enter the matrix: " << endl;
    int *matrix = new int[n * n];
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            cin >> matrix[i * n + j];
        }
    }

    ofstream outfile("cuda.out");
    
    int *sums = new int[n];
    int *d_matrix;
    int *d_sums;
    hipMalloc(&d_matrix, n * n * sizeof(int));
    hipMalloc(&d_sums, n * sizeof(int));
    hipMemcpy(d_matrix, matrix, n * n * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(512);
    dim3 numBlocks((n + blockSize.x - 1) / blockSize.x);

    auto start = std::chrono::high_resolution_clock::now();
    rowsum<<<numBlocks, blockSize>>>(d_matrix, d_sums, n);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;
    // cast it to microseconds
auto elapsed1_casted = std::chrono::duration_cast<std::chrono::microseconds>(elapsed1).count();

     

    outfile << "Row sums: " << endl;
    hipMemcpy(sums, d_sums, n * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; ++i)
    {
        outfile << sums[i] << " ";
    }
    outfile << endl;

    int *njMatrix = new int[n * n];
    int *d_njMatrix;
    hipMalloc(&d_njMatrix, n * n * sizeof(int));
     start = std::chrono::high_resolution_clock::now();
    neighborJoiningMatrix<<<numBlocks, blockSize>>>(d_matrix, d_sums, d_njMatrix, n);
    hipDeviceSynchronize();
     end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;
  auto elapsed2_casted = std::chrono::duration_cast<std::chrono::microseconds>(elapsed2).count();


    hipMemcpy(njMatrix, d_njMatrix, n * n * sizeof(int), hipMemcpyDeviceToHost);
    outfile << "Neighbor joining matrix: " << endl;
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            outfile << njMatrix[i * n + j] << " ";
        }
        outfile << endl;
    }

    int *d_minVal, *d_minIndexI, *d_minIndexJ, *d_delta;
    hipMalloc(&d_minVal, sizeof(int));
    hipMalloc(&d_minIndexI, sizeof(int));
    hipMalloc(&d_minIndexJ, sizeof(int));
    hipMalloc(&d_delta, sizeof(int));
    hipMemcpy(d_minVal, new int, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_minIndexI, new int, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_minIndexJ, new int, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_delta, new int, sizeof(int), hipMemcpyHostToDevice);

    // shared memory
    int sharedMemSize = blockSize.x * sizeof(int);
     start = std::chrono::high_resolution_clock::now();
    findMinIndices<<<numBlocks, blockSize, sharedMemSize>>>(d_njMatrix, d_sums, d_minVal, d_minIndexI, d_minIndexJ, d_delta, n);
    hipDeviceSynchronize();
     end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;
 auto elapsed3_casted = std::chrono::duration_cast<std::chrono::microseconds>(elapsed3).count();


    // print the deltas
    int delta;

    hipMemcpy(&delta, d_delta, sizeof(int), hipMemcpyDeviceToHost);
    outfile << "Delta: " << delta << endl;

    

    //print the timings in another file
   std::ofstream file2("cuda_timing.out");
if(file2.is_open()) {
    //print the casted values
    file2 <<"For rowsum: " <<elapsed1_casted << " Microseconds"<<"\n";
    file2 <<"For NEIGHBORJOINING matrix: "<< elapsed2_casted << " Microseconds" <<"\n";
    file2 << "For MinIndices: "<<elapsed3_casted << "Microseconds"<<"\n";
    file2.close();
} else {
    std::cout << "Unable to open file";
}

    delete[] matrix;
    delete[] sums;
    hipFree(d_matrix);
    hipFree(d_sums);
    hipFree(d_njMatrix);
    outfile.close();

    return 0;
}
