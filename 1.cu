#include <bits/stdc++.h>
#include <numeric>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

__global__ void rowsum(int *d_matrix, int *d_sums, int n)
{
    long block_Idx = blockIdx.x + (gridDim.x) * blockIdx.y + (gridDim.y * gridDim.x) * blockIdx.z;
    long thread_Idx = threadIdx.x + (blockDim.x) * threadIdx.y + (blockDim.y * blockDim.x) * threadIdx.z;
    long block_Capacity = blockDim.x * blockDim.y * blockDim.z;
    long i = block_Idx * block_Capacity + thread_Idx;

    if (i < n)
    {
        d_sums[i] = 0; // Initialize the sum to 0
        for (int j = 0; j < n; ++j)
        {
            atomicAdd(&d_sums[i], d_matrix[i * n + j]);
        }
    }
}

__global__ void neighborJoiningMatrix(int *d_matrix, int *d_rowSums, int *d_njMatrix, int n)
{
    long block_Idx = blockIdx.x + (gridDim.x) * blockIdx.y + (gridDim.y * gridDim.x) * blockIdx.z;
    long thread_Idx = threadIdx.x + (blockDim.x) * threadIdx.y + (blockDim.y * blockDim.x) * threadIdx.z;
    long block_Capacity = blockDim.x * blockDim.y * blockDim.z;
    long arr_Idx = block_Idx * block_Capacity + thread_Idx;

    if (arr_Idx < n * n)
    {
        int i = arr_Idx / n;
        int j = arr_Idx % n;

        if (i == j)
        {
            d_njMatrix[arr_Idx] = 0;
        }
        else
        {
            d_njMatrix[arr_Idx] = (n - 2) * d_matrix[arr_Idx] - d_rowSums[i] - d_rowSums[j];
        }
    }
}

__global__ void findMinAndComputeDelta(int *d_njMatrix, int *d_rowSums, int *d_minVal, int *d_minIndices, int n)
{
    extern __shared__ int sharedData[];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory
    sharedData[tid] = d_njMatrix[index];
    __syncthreads();

    // Reduction to find minimum
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            if (sharedData[tid] > sharedData[tid + s])
            {
                sharedData[tid] = sharedData[tid + s];
                d_minIndices[blockIdx.x] = index + s;
            }
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0)
    {
        d_minVal[blockIdx.x] = sharedData[0];
    }
}

__global__ void computeDelta(int *d_rowSums, int *d_delta, int *d_minIndices, int n)
{
    int index = threadIdx.x;

    if (index == 0)
    {
        d_delta[0] = (d_rowSums[d_minIndices[0]] - d_rowSums[d_minIndices[1]]) / (n - 2);
    }
}

int main()
{
    int n;
    cout << "Enter the size of the matrix: " << endl;
    cin >> n;
    cout << "Enter the matrix: " << endl;
    int *matrix = new int[n * n];
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            cin >> matrix[i * n + j];
        }
    }
    int *sums = new int[n];
    int *d_matrix;
    int *d_sums;
    hipMalloc(&d_matrix, n * n * sizeof(int));
    hipMalloc(&d_sums, n * sizeof(int));
    hipMemcpy(d_matrix, matrix, n * n * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(512);
    dim3 numBlocks((n + blockSize.x - 1) / blockSize.x);

    rowsum<<<numBlocks, blockSize>>>(d_matrix, d_sums, n);
    hipDeviceSynchronize();
    cout << "Row sums: " << endl;
    hipMemcpy(sums, d_sums, n * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; ++i)
    {
        cout << sums[i] << " ";
    }
    cout << endl;

    int *njMatrix = new int[n * n];
    int *d_njMatrix;
    hipMalloc(&d_njMatrix, n * n * sizeof(int));
    neighborJoiningMatrix<<<numBlocks, blockSize>>>(d_matrix, d_sums, d_njMatrix, n);
    hipDeviceSynchronize();
    hipMemcpy(njMatrix, d_njMatrix, n * n * sizeof(int), hipMemcpyDeviceToHost);
    cout << "Neighbor joining matrix: " << endl;
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            cout << njMatrix[i * n + j] << " ";
        }
        cout << endl;
    }

    int *minVal = new int[n - 1];
    int *minIndices = new int[n - 1];
    int *d_minVal;
    int *d_minIndices;
    hipMalloc(&d_minVal, (n - 1) * sizeof(int));
    hipMalloc(&d_minIndices, (n - 1) * sizeof(int));
    findMinAndComputeDelta<<<numBlocks, blockSize, blockSize.x * sizeof(int)>>>(d_njMatrix, d_sums, d_minVal, d_minIndices, n);
    hipDeviceSynchronize();
    hipMemcpy(minVal, d_minVal, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(minIndices, d_minIndices, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
    cout << "Minimum values: " << endl;
    for (int i = 0; i < n - 1; ++i)
    {
        cout << minVal[i] << " ";
    }
    cout << endl;
    cout << "Minimum indices: " << endl;
    for (int i = 0; i < n - 1; ++i)
    {
        cout << minIndices[i] << " ";
    }
    cout << endl;
    

    delete[] matrix;
    delete[] sums;
    hipFree(d_matrix);
    hipFree(d_sums);
    hipFree(d_njMatrix);
    
    return 0;
}